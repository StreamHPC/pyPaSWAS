
#include <hip/hip_runtime.h>
/** Direction definitions for the direction matrix. These are needed for the trace back */
#define NO_DIRECTION ${NO_DIRECTION}
#define STOP_DIRECTION ${STOP_DIRECTION}

#define UPPER_LEFT_DIRECTION ${UP_LEFT_DIRECTION}
#define UPPER_DIRECTION ${UP_DIRECTION}
#define LEFT_DIRECTION ${LEFT_DIRECTION}

/** Direction definitions for affine gap penalty **/
#define MAIN_MATRIX ${UP_LEFT_DIRECTION}
#define I_MATRIX ${UP_DIRECTION}
#define J_MATRIX ${LEFT_DIRECTION}

#define A_DIRECTION (${UP_LEFT_DIRECTION} << 2)
#define B_DIRECTION (${UP_DIRECTION} <<2)
#define C_DIRECTION (${LEFT_DIRECTION} <<2)
#define DIRECTION_MASK 12
#define MATRIX_MASK 3

